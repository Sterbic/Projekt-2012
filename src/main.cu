#include "hip/hip_runtime.h"
#include <algorithm>
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <vector>

#include "Builders.h"
#include "Defines.h"
#include "FASTA.h"
#include "FindAlignment.cuh"
#include "SWutils.h"
#include "Traceback.cuh"

int main(int argc, char *argv[]) {
    printf("### Welcome to SWalign v%s\n\n", VERSION);
    cudaTimer timer;
    timer.start();

    if (argc != 7) {
        printf("Expected 6 input arguments, not %d!\n\n", argc - 1);
        return -1;
    }

    FASTAsequence first(argv[1]);
    FASTAsequence second(argv[2]);

    printf("> Loading input sequences... ");
    if(!first.load() || !second.load())
    	exitWithMsg("An error has occured while loading input sequences.", -1);
    else
    	printf("DONE\n\n");
	
    printf("First sequence of length %d:\n%s\n\n", first.getLength(), first.getSequenceName());
    printf("Second sequence of length %d:\n%s\n\n", second.getLength(), second.getSequenceName());

    printf("> Looking for CUDA capable cards... ");
    CUDAcard bestGpu = findBestDevice();
    safeAPIcall(hipSetDevice(bestGpu.cardNumber), __LINE__);
    printf("DONE\n\n");
    printf("Found %d CUDA capable GPU(s), picked GPU number %d:\n",
    		bestGpu.cardsInSystem, bestGpu.cardNumber + 1);
    printCardInfo(bestGpu);
    printf("\n");

    printf("> Initializing launch configuration... ");
    LaunchConfig config = getLaunchConfig(
    		min(first.getLength(), second.getLength()),
    		bestGpu);
    printf("DONE\n\n");
    printLaunchConfig(config);

    printf("\n> Preparing SWquery... ");
    SWquery query(&first, &second);
    query.prepare(config);
    printf("DONE\n\n");

    scoring values = initScoringValues(argv[3], argv[4], argv[5], argv[6]);

	//###################### finding alignment ##############################
    printf("> Starting alignment process... ");

    alignmentScore *score;
    int scoreSize = sizeof(alignmentScore) * config.blocks * config.threads;
    score = (alignmentScore *) malloc(scoreSize);
    if(score == NULL)
    	exitWithMsg("An error has occured while allocating blockScores array on host.", -1);
    
	alignmentScore *devScore = (alignmentScore *) cudaGetSpaceAndSet(scoreSize, 0);

    GlobalBuffer buffer;
    initGlobalBuffer(&buffer, query.getSecond()->getPaddedLength(), config);

	int D = config.blocks + ceil(((double) query.getFirst()->getPaddedLength())
			/ (ALPHA * config.threads)) - 1;

	safeAPIcall(hipFuncSetCacheConfig(reinterpret_cast<const void*>(shortPhase), hipFuncCachePreferShared), __LINE__);
	safeAPIcall(hipFuncSetCacheConfig(reinterpret_cast<const void*>(longPhase), hipFuncCachePreferShared), __LINE__);
	
	safeAPIcall(hipBindTexture(
			NULL,
			texSecond,
			query.getDevSecond(),
			query.getSecond()->getPaddedLength()
			), __LINE__);

	RowBuilder rowBuilder(
			query.getFirst()->getPaddedLength(),
			query.getSecond()->getPaddedLength(),
			&config
			);

    cudaTimer kernelTimer;
    kernelTimer.start();

    for(int dk = 0; dk < D + config.blocks; ++dk) {
    	shortPhase<<<config.blocks, config.threads, config.sharedMemSize>>>(
    			dk,
    			buffer.hBuffer,
    			buffer.vBuffer,
    			query.getDevFirst(),
    			query.getFirst()->getPaddedLength(),
    			query.getSecond()->getPaddedLength(),
    			values,
    			devScore
    			);

    	rowBuilder.dumpShort(buffer.hBuffer.up, dk);

		longPhase<<<config.blocks, config.threads, config.sharedMemSize>>>(
				dk,
    			buffer.hBuffer,
    			buffer.vBuffer,
    			query.getDevFirst(),
    			query.getFirst()->getPaddedLength(),
    			query.getSecond()->getPaddedLength(),
				values,
				devScore
				);

		rowBuilder.dumpLong(buffer.hBuffer.up, dk);
    }
    
    kernelTimer.stop();

    safeAPIcall(hipMemcpy(score, devScore, scoreSize, hipMemcpyDeviceToHost), __LINE__);
	alignmentScore max = getMaxScore(score, config.blocks * config.threads);

	timer.stop();

    printf("DONE\n\n");

    double gcups = first.getLength() / 1e6 * second.getLength() / (timer.getElapsedTimeMillis());
    printf("\t>Kernel executed in %f s\n", kernelTimer.getElapsedTimeMillis() / 1000);
    printf("\t>Application executed in %f s\n", timer.getElapsedTimeMillis() / 1000);
    printf("\t>Cell updates per second: %lf GCUPS\n", gcups);
    printf("\t>Alignment score: %d at [%d, %d]\n\n", max.score, max.row + 1, max.column + 1);

    safeAPIcall(hipUnbindTexture(texSecond), __LINE__);

    safeAPIcall(hipFree(devScore), __LINE__);

    freeGlobalBuffer(&buffer);

    free(score);

	//######################## traceback #################################
    HorizontalBuffer hBuffer;
    int2 *hBufferUp = (int2 *) malloc(sizeof(int2) * rowBuilder.getRowHeight());
    if(hBufferUp == NULL)
    	exitWithMsg("Allocation error hBufferUp", -1);

    for(int i = 0; i < rowBuilder.getRowHeight(); i++)
    	hBufferUp[i].x = hBufferUp[i].y = -max.score;

    hBuffer.up = (int2 *) cudaGetDeviceCopy(hBufferUp, sizeof(int2) * rowBuilder.getRowHeight());

    int widthOffset = 0;
    int heightOffset = 0;
    int specialRowIndex = (max.row / rowBuilder.getRowHeight()) * rowBuilder.getRowHeight();
    int chunkSize = rowBuilder.getRowHeight();
    char fileName[50];

    LaunchConfig traceback = getLaunchConfig(chunkSize, bestGpu);
    VerticalBuffer vBuffer;
    initVerticalBuffer(&vBuffer, traceback);

    int paddedChunkHeight = chunkSize;
    if(chunkSize % traceback.blocks != 0)
    	paddedChunkHeight += traceback.blocks - (chunkSize % traceback.blocks);

    int paddedChunkWidth = paddedChunkHeight;
    if(paddedChunkWidth % 4 != 0)
    	paddedChunkWidth += 4 - (paddedChunkWidth % 4);

    char *devRow = (char *) cudaGetSpaceAndSet(paddedChunkWidth * sizeof(char), 0);
    char *devColumn = (char *) cudaGetSpaceAndSet(paddedChunkHeight * sizeof(char), 0);

    int2 *vBusOut = (int2 *) malloc(chunkSize * sizeof(int2));
    memset(vBusOut, -1, chunkSize * sizeof(int2));
    int2 *devVBusOut = (int2 *) cudaGetSpaceAndSet(paddedChunkWidth * sizeof(int2), 0);
    char pad[240];
    memset(pad, STAGE_2_PADDING, 240);

    char *firstReversed = query.getFirst()->getReversedSequence(max.row);
    char *secondReversed = query.getSecond()->getReversedSequence(max.column);

    bool gap = false;

    std::vector<TracebackScore> crosspoints;

    TracebackScore maxTrace;
    maxTrace.score = max.score;
    maxTrace.column = max.column;
    maxTrace.row = max.row;
    maxTrace.gap = gap;

    crosspoints.push_back(maxTrace);

    //printf("\nSR size = %ld\n", query.getSecond()->getPaddedLength() * sizeof(int2));
	int2 *specialRow = (int2 *) malloc(query.getSecond()->getPaddedLength() * sizeof(int2));
	if(specialRow == NULL)
		exitWithMsg("Error allocating special row.", -1);

    D = traceback.blocks + ceil(((double) std::max(paddedChunkHeight, paddedChunkWidth))
    			/ (ALPHA * traceback.threads)) - 1;

	int readOffset = 0;
    while(maxTrace.score > min(rowBuilder.getRowHeight(), maxTrace.row - specialRowIndex + 1) * values.match) {

		memset(fileName, 0, 50);
		sprintf(fileName, "temp/row_%d", specialRowIndex);
		//printf("%s\n", fileName);
		FILE *f = fopen(fileName, "rb");
		if(f == NULL)
			exitWithMsg("Error opening special row file.", -1);

		fread(specialRow, sizeof(int2), query.getSecond()->getPaddedLength(), f);
		fclose(f);

		int getVertical = min(chunkSize, maxTrace.row - specialRowIndex + 1);

		safeAPIcall(hipMemcpy(devColumn, firstReversed + heightOffset,
				getVertical * sizeof(char), hipMemcpyHostToDevice), __LINE__);

		//printf("Padded H = %d, Padded W = %d\n", paddedChunkHeight, paddedChunkWidth);
		for(int i = getVertical; i < paddedChunkHeight - getVertical; i += 240) {
			//printf("i = %d ", i);
			safeAPIcall(hipMemcpy(devColumn + i, pad, min(paddedChunkHeight - i, 240) * sizeof(char),
					hipMemcpyHostToDevice), __LINE__);
		}
		//printf("getVertical = %d\n", getVertical);

		while(widthOffset < maxTrace.column) {

			int getNum = min(min(chunkSize, getVertical), maxTrace.column - widthOffset + 1);
			printf("getNum = %d, offset = %d\n", getNum, widthOffset + readOffset);
			safeAPIcall(hipMemcpy(devRow, secondReversed + widthOffset + readOffset,
					getNum * sizeof(char), hipMemcpyHostToDevice), __LINE__);

			for(int i = getNum; i < paddedChunkWidth - getNum; i += 240) {
				//printf("i = %d ", i);
				safeAPIcall(hipMemcpy(devRow + i, pad,
						min(paddedChunkWidth - i, 240) * sizeof(char), hipMemcpyHostToDevice), __LINE__);
			}

			//printf("iter = %d\n", D + traceback.blocks);
			for(int dk = 0; dk < D + traceback.blocks; ++dk) {
				tracebackShort<<<traceback.blocks, traceback.threads, traceback.sharedMemSize>>>(
							dk,
							hBuffer,
							vBuffer,
							devRow,
							paddedChunkWidth,
							devColumn,
							paddedChunkHeight,
							values,
							devVBusOut,
							gap
							);

				tracebackLong<<<traceback.blocks, traceback.threads, traceback.sharedMemSize>>>(
							dk,
							hBuffer,
							vBuffer,
							devRow,
							paddedChunkWidth,
							devColumn,
							paddedChunkHeight,
							values,
							devVBusOut,
							gap
							);
			}

	/*		int2 *vBusPadded = (int2 *) malloc(paddedChunkWidth * sizeof(int2));
			safeAPIcall(hipMemcpy(vBusPadded, devVBusOut,
					paddedChunkWidth * sizeof(int2), hipMemcpyDeviceToHost), __LINE__);
			FILE *tmp1 = fopen("temp/vbusout1.txt", "a");
			for(int i = 0; i < paddedChunkWidth; i++) {
				fprintf(tmp1, "%d %d\n", (vBusPadded + i)->x, (vBusPadded + i)->x);
			}
			fclose(tmp1);
			free(vBusPadded); */

			safeAPIcall(hipMemcpy(vBusOut, devVBusOut + paddedChunkWidth - getNum, // po meni, tu je getNum, a ne chunkSize
					getNum * sizeof(int2), hipMemcpyDeviceToHost), __LINE__);

			FILE *tmp = fopen("temp/vbusout.txt", "a");
			for(int i = 0; i < getNum; i++) {
				fprintf(tmp, "%d %d\n", (vBusOut + i)->x, (vBusOut + i)->y);
			}
			fclose(tmp);

			/*
			TracebackScore getTracebackScore(scoring values, int row, int cols,
			int2 *vBusOut, int2 *specialRow, int targetScore, int absColIdx);
			*/
			
			TracebackScore tracebackScore = getTracebackScore(
					values, specialRowIndex - 1, getNum, vBusOut,
					specialRow + maxTrace.column - widthOffset - getNum - 1, maxTrace.score, maxTrace.column - widthOffset);
			//printf("\nTrace [%d, %d] = %d\n", tracebackScore.row, tracebackScore.column, tracebackScore.score);

			if(tracebackScore.column != -1) {
				readOffset += maxTrace.column - tracebackScore.column;
				maxTrace.score = tracebackScore.score;
				maxTrace.column = tracebackScore.column;
				maxTrace.row = tracebackScore.row;
				maxTrace.gap = tracebackScore.gap;
				gap = tracebackScore.gap;

				printf("Crosspoint [%d, %d] = %d\n", maxTrace.row, maxTrace.column, maxTrace.score);
				crosspoints.push_back(maxTrace);

				specialRowIndex -= rowBuilder.getRowHeight();
				widthOffset = 0; // ako smo nasli crosspoint
				heightOffset += getVertical;
				
				safeAPIcall(hipMemcpy(hBuffer.up, hBufferUp, sizeof(int2) * rowBuilder.getRowHeight(),
						hipMemcpyHostToDevice), __LINE__);

				break;
			}
			else {
				widthOffset += getNum; 
			}
				// ako nismo nasli crosspoint, pomicemo se u stranu za onoliko koliko smo elemenata obradili
		}
    }

    //########################### finding alignment start point ###################################

    printf("\nStarting last with target score %d\n", maxTrace.score);

    char padLastRows[240];
    memset(padLastRows, STAGE_2_PADDING_LAST_ROWS, 240);

    int lastSize = traceback.blocks * traceback.threads * sizeof(TracebackScore);
    TracebackScore *last = (TracebackScore *) malloc(lastSize);
    if(last == NULL)
    	exitWithMsg("Allocation error for traceback last", -1);
    TracebackScore *devLast = (TracebackScore *) cudaGetSpaceAndSet(lastSize, -1);

    if(maxTrace.score != 0) {
    	int getVertical = min(chunkSize, maxTrace.row + 1);
    	printf("getv = %d", getVertical);
    	safeAPIcall(hipMemcpy(devColumn, firstReversed + heightOffset,
    			getVertical * sizeof(char), hipMemcpyHostToDevice), __LINE__);

    	for(int i = getVertical; i < paddedChunkHeight - getVertical; i += 240)
			safeAPIcall(hipMemcpy(devColumn + i, padLastRows,
					min(paddedChunkWidth - i, 240) * sizeof(char), hipMemcpyHostToDevice), __LINE__);

    	bool found = false;
    	while(widthOffset < maxTrace.column) {
    		int getNum = min(chunkSize, max.column - widthOffset + 1);
			safeAPIcall(hipMemcpy(devRow, secondReversed + widthOffset,
					getNum * sizeof(char), hipMemcpyHostToDevice), __LINE__);

			for(int i = getNum; i < paddedChunkWidth - getNum; i += 240)
				safeAPIcall(hipMemcpy(devRow + i, pad,
						min(paddedChunkWidth - i, 240) * sizeof(char), hipMemcpyHostToDevice), __LINE__);

			for(int dk = 0; dk < D + traceback.blocks; dk++) {
				tracebackLastShort<<<traceback.blocks, traceback.threads, traceback.sharedMemSize>>>(
							dk,
							hBuffer,
							vBuffer,
							devRow,
							paddedChunkWidth,
							devColumn,
							paddedChunkHeight,
							values,
							gap,
							devLast,
							maxTrace.score
							);

				safeAPIcall(hipMemcpy(last, devLast, lastSize, hipMemcpyDeviceToHost), __LINE__);
				printf("Short %d\n", dk);
				for(int i = 0; i < traceback.blocks * traceback.threads; i++) {
					if(last[i].score != -1) {
						TracebackScore lastScore;
						lastScore.score = last[i].score;
						lastScore.row = maxTrace.row - last[i].row;
						lastScore.column = maxTrace.column - last[i].column - widthOffset;
						lastScore.gap = false;

						printf("last[i].row = %d, maxTrace.row = %d\n", last[i].row, maxTrace.row);
						printf("Found last in short: [%d, %d] = %d\n", lastScore.row,
								lastScore.column, lastScore.score);
						crosspoints.push_back(lastScore);
						found = true;
						break;
					}
				}

				if(found) break;

				tracebackLastLong<<<traceback.blocks, traceback.threads, traceback.sharedMemSize>>>(
							dk,
							hBuffer,
							vBuffer,
							devRow,
							paddedChunkWidth,
							devColumn,
							paddedChunkHeight,
							values,
							gap,
							devLast,
							maxTrace.score
							);

				safeAPIcall(hipMemcpy(last, devLast, lastSize, hipMemcpyDeviceToHost), __LINE__);
				printf("Long %d\n", dk);
				for(int i = 0; i < traceback.blocks * traceback.threads; i++) {
					if(last[i].score != -1) {
						TracebackScore lastScore;
						lastScore.score = last[i].score;
						lastScore.row = maxTrace.row - last[i].row;
						lastScore.column = maxTrace.column - last[i].column - widthOffset;
						lastScore.gap = false;

						printf("last[i].row = %d, maxTrace.row = %d, wo = %d\n", last[i].row, maxTrace.row, widthOffset);
						printf("last[i].col = %d, maxTrace.col = %d\n", last[i].column, maxTrace.column);

						printf("Found last in long: [%d, %d] = %d\n", lastScore.row,
								lastScore.column, lastScore.score);
						crosspoints.push_back(lastScore);
						found = true;
						break;
					}
				}

				if(found) break;
			}

			if(found) break;
			widthOffset += getNum;
    	}
    }

    printf("END\n");

    safeAPIcall(hipFree(devColumn), __LINE__);
    safeAPIcall(hipFree(devRow), __LINE__);
    safeAPIcall(hipFree(devVBusOut), __LINE__);
    safeAPIcall(hipFree(devLast), __LINE__);

    free(specialRow);
    free(last);

    safeAPIcall(hipFree(hBuffer.up), __LINE__);
    freeVerticalBuffer(&vBuffer);

    return 0;
}
