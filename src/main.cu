#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <algorithm>
#include <vector>

#include "Defines.h"
#include "FASTA.h"
#include "SWutils.h"
#include "Builders.h"
#include "FindAlignment.cuh"
#include "Traceback.cuh"

int main(int argc, char *argv[]) {
    printf("### Welcome to SWalign v%s\n\n", VERSION);
    cudaTimer timer;
    timer.start();

    if (argc != 7) {
        printf("Expected 6 input arguments, not %d!\n\n", argc - 1);
        return -1;
    }

    FASTAsequence first(argv[1]);
    FASTAsequence second(argv[2]);

    printf("> Loading input sequences... ");
    if(!first.load() || !second.load())
    	exitWithMsg("An error has occured while loading input sequences.", -1);
    else
    	printf("DONE\n\n");
	
    printf("First sequence of length %d:\n%s\n\n", first.getLength(), first.getSequenceName());
    printf("Second sequence of length %d:\n%s\n\n", second.getLength(), second.getSequenceName());

    printf("> Looking for CUDA capable cards... ");
    CUDAcard bestGpu = findBestDevice();
    safeAPIcall(hipSetDevice(bestGpu.cardNumber), __LINE__);
    printf("DONE\n\n");
    printf("Found %d CUDA capable GPU(s), picked GPU number %d:\n",
    		bestGpu.cardsInSystem, bestGpu.cardNumber + 1);
    printCardInfo(bestGpu);
    printf("\n");

    printf("> Initializing launch configuration... ");
    LaunchConfig config = getLaunchConfig(
    		min(first.getLength(), second.getLength()),
    		bestGpu);
    printf("DONE\n\n");
    printLaunchConfig(config);

    printf("\n> Preparing SWquery... ");
    SWquery query(&first, &second);
    query.prepare(config);
    printf("DONE\n\n");

    scoring values = initScoringValues(argv[3], argv[4], argv[5], argv[6]);

    printf("> Starting alignment process... ");

    alignmentScore *score;
    int scoreSize = sizeof(alignmentScore) * config.blocks * config.threads;
    score = (alignmentScore *) malloc(scoreSize);
    if(score == NULL)
    	exitWithMsg("An error has occured while allocating blockScores array on host.", -1);
    
	alignmentScore *devScore = (alignmentScore *) cudaGetSpaceAndSet(scoreSize, 0);

    GlobalBuffer buffer;
    initGlobalBuffer(&buffer, query.getSecond()->getPaddedLength(), config);

	int D = config.blocks + ceil(((double) query.getFirst()->getPaddedLength())
			/ (ALPHA * config.threads)) - 1;

	safeAPIcall(hipFuncSetCacheConfig(reinterpret_cast<const void*>(shortPhase), hipFuncCachePreferShared), __LINE__);
	safeAPIcall(hipFuncSetCacheConfig(reinterpret_cast<const void*>(longPhase), hipFuncCachePreferShared), __LINE__);
	
	safeAPIcall(hipBindTexture(
			NULL,
			texSecond,
			query.getDevSecond(),
			query.getSecond()->getPaddedLength()
			), __LINE__);

	RowBuilder rowBuilder(
			query.getFirst()->getPaddedLength(),
			query.getSecond()->getPaddedLength(),
			&config
			);

    cudaTimer kernelTimer;
    kernelTimer.start();

    for(int dk = 0; dk < D + config.blocks; dk++) {
    	shortPhase<<<config.blocks, config.threads, config.sharedMemSize>>>(
    			dk,
    			buffer.hBuffer,
    			buffer.vBuffer,
    			query.getDevFirst(),
    			query.getFirst()->getPaddedLength(),
    			query.getSecond()->getPaddedLength(),
    			values,
    			devScore
    			);

    	rowBuilder.dumpShort(buffer.hBuffer.up, dk);

		longPhase<<<config.blocks, config.threads, config.sharedMemSize>>>(
				dk,
    			buffer.hBuffer,
    			buffer.vBuffer,
    			query.getDevFirst(),
    			query.getFirst()->getPaddedLength(),
    			query.getSecond()->getPaddedLength(),
				values,
				devScore
				);

		rowBuilder.dumpLong(buffer.hBuffer.up, dk);
    }
    
    kernelTimer.stop();

    safeAPIcall(hipMemcpy(score, devScore, scoreSize, hipMemcpyDeviceToHost), __LINE__);
	alignmentScore max = getMaxScore(score, config.blocks * config.threads);

	timer.stop();

    printf("DONE\n\n");

    double gcups = first.getLength() / 1e6 * second.getLength() / (timer.getElapsedTimeMillis());
    printf("\t>Kernel executed in %f s\n", kernelTimer.getElapsedTimeMillis() / 1000);
    printf("\t>Application executed in %f s\n", timer.getElapsedTimeMillis() / 1000);
    printf("\t>Cell updates per second: %lf GCUPS\n", gcups);
    printf("\t>Alignment score: %d at [%d, %d]\n\n", max.score, max.row + 1, max.column + 1);

    safeAPIcall(hipUnbindTexture(texSecond), __LINE__);

    safeAPIcall(hipFree(devScore), __LINE__);

    freeGlobalBuffer(&buffer);

    free(score);

    HorizontalBuffer hBuffer;
    hBuffer.up = (int2 *) cudaGetSpaceAndSet(sizeof(int2) * rowBuilder.getRowHeight(), -max.score);
    VerticalBuffer vBuffer;
    initVerticalBuffer(&vBuffer, config);

    int widthOffset = 0;
    int heigthOffset = 0;
    int specialRowIndex = (max.row / rowBuilder.getRowHeight()) * rowBuilder.getRowHeight();
    int chunkSize = rowBuilder.getRowHeight();
    char fileName[50];

    LaunchConfig traceback = getLaunchConfig(chunkSize, bestGpu);
    int paddedChunkHeight = chunkSize;
    if(chunkSize % traceback.blocks != 0)
    	paddedChunkHeight += traceback.blocks - (chunkSize % traceback.blocks);

    int paddedChunkWidth = paddedChunkHeight;
    if(paddedChunkWidth % 4 != 0)
    	paddedChunkWidth += 4 - (paddedChunkHeight % 4);

    char *devRow = (char *) cudaGetSpaceAndSet(paddedChunkWidth * sizeof(char), 0);
    char *devColumn = (char *) cudaGetSpaceAndSet(paddedChunkHeight * sizeof(char), 0);

    int2 *vBusOut = (int2 *) malloc(chunkSize * sizeof(int2));
    int2 *devVBusOut = (int2 *) cudaGetSpaceAndSet(paddedChunkWidth * sizeof(int2), 0);
    char pad[240];
    memset(pad, STAGE_2_PADDING, 240);

    char *firstReversed = query.getFirst()->getReversedSequence(max.row);
    char *secondReversed = query.getSecond()->getReversedSequence(max.column);

    bool gap = false;

    std::vector<TracebackScore> crosspoints;

    TracebackScore maxTrace;
    maxTrace.score = max.score;
    maxTrace.column = max.column;
    maxTrace.row = max.row;
    maxTrace.gap = gap;

    crosspoints.push_back(maxTrace);

	int2 *specialRow = (int2 *) malloc(query.getSecond()->getPaddedLength() * sizeof(int2));
	if(specialRow == NULL)
		exitWithMsg("Error allocating special row.", -1);

    D = traceback.blocks + ceil(((double) std::max(paddedChunkHeight, paddedChunkWidth))
    			/ (ALPHA * traceback.threads)) - 1;

    while(maxTrace.score >= rowBuilder.getRowHeight() * values.match) {

		memset(fileName, 0, 50);
		sprintf(fileName, "temp/row_%d", specialRowIndex);
		printf("%s %d\n", fileName, rowBuilder.getRowHeight());
		FILE *f = fopen(fileName, "rb");
		if(f == NULL)
			exitWithMsg("Error opening special row file.", -1);

		fread(specialRow, sizeof(int2), query.getSecond()->getPaddedLength(), f);
		fclose(f);

		int getVertical = min(chunkSize, maxTrace.row - specialRowIndex);
		safeAPIcall(hipMemcpy(devColumn, firstReversed + heigthOffset,
				getVertical, hipMemcpyHostToDevice), __LINE__);

		printf("Padded H = %d\n", paddedChunkHeight);
		for(int i = getVertical; i < paddedChunkHeight - getVertical; i += 240) {
			printf("i = %d ", i);
			safeAPIcall(hipMemcpy(devColumn + i, pad, min(paddedChunkHeight - i, 240), hipMemcpyHostToDevice), __LINE__);
		}

		while(widthOffset < max.column) {

			int getNum = min(chunkSize, max.column - widthOffset);
			safeAPIcall(hipMemcpy(devRow, secondReversed + widthOffset,
					getNum, hipMemcpyHostToDevice), __LINE__);

			for(int i = getNum; i < paddedChunkWidth - getNum; i += 240)
				safeAPIcall(hipMemcpy(devRow + i, pad,
						min(paddedChunkWidth - i, 240), hipMemcpyHostToDevice), __LINE__);

			for(int dk = 0; dk < D + traceback.blocks; dk++) {
				tracebackShort<<<traceback.blocks, traceback.threads, traceback.sharedMemSize>>>(
							dk,
							hBuffer,
							vBuffer,
							devRow,
							paddedChunkWidth,
							devColumn,
							paddedChunkHeight,
							values,
							devVBusOut,
							gap
							);

				tracebackLong<<<traceback.blocks, traceback.threads, traceback.sharedMemSize>>>(
							dk,
							hBuffer,
							vBuffer,
							devRow,
							paddedChunkWidth,
							devColumn,
							paddedChunkHeight,
							values,
							devVBusOut,
							gap
							);
			}

			safeAPIcall(hipMemcpy(vBusOut, devVBusOut + paddedChunkWidth - chunkSize,
					chunkSize, hipMemcpyDeviceToHost), __LINE__);



			TracebackScore tracebackScore = getTracebackScore(
					values, gap, specialRowIndex, chunkSize, chunkSize, vBusOut, specialRow);

			maxTrace.score -= tracebackScore.score;
			maxTrace.column -= tracebackScore.column;
			maxTrace.row -= tracebackScore.row;
			maxTrace.gap = tracebackScore.gap;
			gap = tracebackScore.gap;

			printf("Crosspoint [%d, %d] = %d", maxTrace.row, maxTrace.column, maxTrace.score);
			crosspoints.push_back(maxTrace);

			specialRowIndex -= rowBuilder.getRowHeight();
			widthOffset += chunkSize;
			heigthOffset += getVertical;
		}
    }

    if(maxTrace.score != 0) {
    	safeAPIcall(hipMemcpy(devColumn, firstReversed + heigthOffset,
    			chunkSize, hipMemcpyHostToDevice), __LINE__);

    	for(int i = chunkSize; i < paddedChunkHeight - chunkSize; i += 240)
			safeAPIcall(hipMemcpy(devColumn + i, pad,
					min(paddedChunkWidth - i, 240), hipMemcpyHostToDevice), __LINE__);

    	while(widthOffset < max.column) {
    		int getNum = min(chunkSize, max.column - widthOffset);
			safeAPIcall(hipMemcpy(devRow, secondReversed + widthOffset,
					getNum, hipMemcpyHostToDevice), __LINE__);

			for(int i = getNum; i < paddedChunkWidth - getNum; i += 240)
				safeAPIcall(hipMemcpy(devRow + i, pad,
						min(paddedChunkWidth - i, 240), hipMemcpyHostToDevice), __LINE__);

			TracebackScore last;
			last.score = maxTrace.score;
			last.column = -10;
			last.row = -10;
			last.gap = false;

			TracebackScore *devLast = (TracebackScore *) cudaGetDeviceCopy(&last, sizeof(TracebackScore));

			for(int dk = 0; dk < D + traceback.blocks; dk++) {
				tracebackLastShort<<<traceback.blocks, traceback.threads, traceback.sharedMemSize>>>(
							dk,
							hBuffer,
							vBuffer,
							devRow,
							paddedChunkWidth,
							devColumn,
							paddedChunkHeight,
							values,
							gap,
							devLast
							);

				safeAPIcall(hipMemcpy(&last, devLast, sizeof(TracebackScore), hipMemcpyDeviceToHost), __LINE__);
				if(last.row != -10) break;

				tracebackLastLong<<<traceback.blocks, traceback.threads, traceback.sharedMemSize>>>(
							dk,
							hBuffer,
							vBuffer,
							devRow,
							paddedChunkWidth,
							devColumn,
							paddedChunkHeight,
							values,
							gap,
							devLast
							);

				safeAPIcall(hipMemcpy(&last, devLast, sizeof(TracebackScore), hipMemcpyDeviceToHost), __LINE__);
				if(last.row != -10) break;
			}

			crosspoints.push_back(last);
    	}
    }

    safeAPIcall(hipFree(devColumn), __LINE__);
    safeAPIcall(hipFree(devRow), __LINE__);
    safeAPIcall(hipFree(devVBusOut), __LINE__);

    free(specialRow);

    safeAPIcall(hipFree(hBuffer.up), __LINE__);
    freeVerticalBuffer(&vBuffer);

    return 0;
}
