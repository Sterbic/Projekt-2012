#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

#include "Defines.h"
#include "FASTA.h"
#include "SWutils.h"
#include "SWkernel.cuh"

__global__ void shortPhase(
		int dk,
		HorizontalBuffer hbuffer,
		VerticalBuffer vbuffer,
		char *first,
		int firstLength,
		char *second,
		int secondLength,
		scoring values,
		alignmentScore *score
		) {

	extern __shared__ int2 iHbuffer[];

	int i = getRow(dk);
	int j = getColumn(secondLength);

	if(j < 0) {
		i -= ALPHA * gridDim.x * blockDim.x;
		j += secondLength;
	}

	if(threadIdx.x == 0)
		iHbuffer[0] = hbuffer.up[j];

	char4 rowBuffer;
	getRowBuffer(i, first, &rowBuffer);

	K iBuffer;
	if(i >= 0 && i < firstLength)
		initK(&iBuffer, i, j, &hbuffer, &vbuffer);

/*	if(threadIdx.x == tid && blockIdx.x == bl) {
		printf("blockIdx.x = %d, threadIdx.x = %d, i = %d, j = %d\n", blockIdx.x, threadIdx.x, i, j);
		printK(&iBuffer);
		printBuffers(&hbuffer, &vbuffer, iHbuffer, secondLength);
	}*/

	int scoreIndex = threadIdx.x + blockIdx.x * blockDim.x;

	for(int innerDiagonal = 0; innerDiagonal < blockDim.x; innerDiagonal++) {

		__syncthreads();

		if(i >= 0 && i < firstLength) {
			int matchMismatch = values.mismatch;
			if(rowBuffer.w == second[j])
				matchMismatch = values.match;

			iBuffer.curr0.y = max(iBuffer.left0.y + values.extension, iBuffer.left0.x + values.first);
			iBuffer.curr0.z = max(iBuffer.up.y + values.extension, iBuffer.up.x + values.first);
			iBuffer.curr0.x = max(max(0, iBuffer.curr0.y), max(iBuffer.curr0.z, iBuffer.diagonal + matchMismatch));

			if(iBuffer.curr0.x > score[scoreIndex].score) {
				score[scoreIndex].score = iBuffer.curr0.x;
				score[scoreIndex].row = i;
				score[scoreIndex].column = j;
			}

			matchMismatch = values.mismatch;
			if(rowBuffer.x == second[j])
				matchMismatch = values.match;

			iBuffer.curr1.y = max(iBuffer.left1.y + values.extension, iBuffer.left1.x + values.first);
			iBuffer.curr1.z = max(iBuffer.curr0.z + values.extension, iBuffer.curr0.x + values.first);
			iBuffer.curr1.x = max(max(0, iBuffer.curr1.y), max(iBuffer.curr1.z, iBuffer.left0.x + matchMismatch));

			if(iBuffer.curr1.x > score[scoreIndex].score) {
				score[scoreIndex].score = iBuffer.curr1.x;
				score[scoreIndex].row = i + 1;
				score[scoreIndex].column = j;
			}

			matchMismatch = values.mismatch;
			if(rowBuffer.y == second[j])
				matchMismatch = values.match;

			iBuffer.curr2.y = max(iBuffer.left2.y + values.extension, iBuffer.left2.x + values.first);
			iBuffer.curr2.z = max(iBuffer.curr1.z + values.extension, iBuffer.curr1.x + values.first);
			iBuffer.curr2.x = max(max(0, iBuffer.curr2.y), max(iBuffer.curr2.z, iBuffer.left1.x + matchMismatch));

			if(iBuffer.curr2.x > score[scoreIndex].score) {
				score[scoreIndex].score = iBuffer.curr2.x;
				score[scoreIndex].row = i + 2;
				score[scoreIndex].column = j;
			}

			matchMismatch = values.mismatch;
			if(rowBuffer.z == second[j])
				matchMismatch = values.match;

			iBuffer.curr3.y = max(iBuffer.left3.y + values.extension, iBuffer.left3.x + values.first);
			iBuffer.curr3.z = max(iBuffer.curr2.z + values.extension, iBuffer.curr2.x + values.first);
			iBuffer.curr3.x = max(max(0, iBuffer.curr3.y), max(iBuffer.curr3.z, iBuffer.left2.x + matchMismatch));

			if(iBuffer.curr3.x > score[scoreIndex].score) {
				score[scoreIndex].score = iBuffer.curr3.x;
				score[scoreIndex].row = i + 3;
				score[scoreIndex].column = j;
			}

			if(threadIdx.x < blockDim.x - 1) {
				iHbuffer[threadIdx.x + 1].x = iBuffer.curr3.x;
				iHbuffer[threadIdx.x + 1].y = iBuffer.curr3.z;
			}
			else {
				hbuffer.up[j].x = iBuffer.curr3.x;
				hbuffer.up[j].y = iBuffer.curr3.z;
			}
		}

		j++;

		__syncthreads();

	/*	if(threadIdx.x == tid && blockIdx.x == bl)
			printBuffers(&hbuffer, &vbuffer, iHbuffer, secondLength);*/

		if(j == secondLength) {
			j = 0;
			i += gridDim.x * ALPHA * blockDim.x;
			getRowBuffer(i, first, &rowBuffer);
			initK(&iBuffer, i, j, iHbuffer, &vbuffer);
		}
		else {
			int2 newUp;
			if(threadIdx.x > 0)
				newUp = iHbuffer[threadIdx.x];
			else
				newUp = hbuffer.up[j];

			pushForwardK(&iBuffer, newUp);
		}

/*		if(threadIdx.x == tid && blockIdx.x == bl) {
			printf("blockIdx.x = %d, threadIdx.x = %d, i = %d, j = %d\n", blockIdx.x, threadIdx.x, i, j);
			printK(&iBuffer);
			printBuffers(&hbuffer, &vbuffer, iHbuffer, secondLength);
		}*/

		__syncthreads();
	}

	if (i >= 0 && i < firstLength) {
		updateVerticalBuffer(&iBuffer, &vbuffer, i);
		if(threadIdx.x < blockDim.x -1)
			hbuffer.up[j - 1] = iHbuffer[threadIdx.x + 1];
	}

/*	if(threadIdx.x == tid && blockIdx.x == bl) {
		printBuffers(&hbuffer, &vbuffer, iHbuffer, secondLength);
	}*/
}

__global__ void longPhase(
		int dk,
		HorizontalBuffer hbuffer,
		VerticalBuffer vbuffer,
		char *first,
		int firstLength,
		char *second,
		int secondLength,
		scoring values,
		alignmentScore *score
		) {

	extern __shared__ int2 iHbuffer[];

	int C = secondLength / gridDim.x;

	int i = getRow(dk);
	int j = getColumn(secondLength) + blockDim.x;

	char4 rowBuffer;
	getRowBuffer(i, first, &rowBuffer);

	K iBuffer;
	if(i >= 0 && i < firstLength)
		initK(&iBuffer, i, j, &hbuffer, &vbuffer);

/*	if(threadIdx.x == tid && blockIdx.x == bl) {
		printf("blockIdx.x = %d, threadIdx.x = %d, i = %d, j = %d\n", blockIdx.x, threadIdx.x, i, j);
		printK(&iBuffer);
		printBuffers(&hbuffer, &vbuffer, iHbuffer, secondLength);
	} */

	int scoreIndex = threadIdx.x + blockIdx.x * blockDim.x;

	__syncthreads();

	for(int innerDiagonal = blockDim.x; innerDiagonal < C; innerDiagonal++) {
		if(i >= 0 && i < firstLength) {
			int matchMismatch = values.mismatch;
			if(rowBuffer.w == second[j])
				matchMismatch = values.match;

			iBuffer.curr0.y = max(iBuffer.left0.y + values.extension, iBuffer.left0.x + values.first);
			iBuffer.curr0.z = max(iBuffer.up.y + values.extension, iBuffer.up.x + values.first);
			iBuffer.curr0.x = max(max(0, iBuffer.curr0.y), max(iBuffer.curr0.z, iBuffer.diagonal + matchMismatch));

			if(iBuffer.curr0.x > score[scoreIndex].score) {
				score[scoreIndex].score = iBuffer.curr0.x;
				score[scoreIndex].row = i;
				score[scoreIndex].column = j;
			}

			matchMismatch = values.mismatch;
			if(rowBuffer.x == second[j])
				matchMismatch = values.match;

			iBuffer.curr1.y = max(iBuffer.left1.y + values.extension, iBuffer.left1.x + values.first);
			iBuffer.curr1.z = max(iBuffer.curr0.z + values.extension, iBuffer.curr0.x + values.first);
			iBuffer.curr1.x = max(max(0, iBuffer.curr1.y), max(iBuffer.curr1.z, iBuffer.left0.x + matchMismatch));

			if(iBuffer.curr1.x > score[scoreIndex].score) {
				score[scoreIndex].score = iBuffer.curr1.x;
				score[scoreIndex].row = i + 1;
				score[scoreIndex].column = j;
			}

			matchMismatch = values.mismatch;
			if(rowBuffer.y == second[j])
				matchMismatch = values.match;

			iBuffer.curr2.y = max(iBuffer.left2.y + values.extension, iBuffer.left2.x + values.first);
			iBuffer.curr2.z = max(iBuffer.curr1.z + values.extension, iBuffer.curr1.x + values.first);
			iBuffer.curr2.x = max(max(0, iBuffer.curr2.y), max(iBuffer.curr2.z, iBuffer.left1.x + matchMismatch));

			if(iBuffer.curr2.x > score[scoreIndex].score) {
				score[scoreIndex].score = iBuffer.curr2.x;
				score[scoreIndex].row = i + 2;
				score[scoreIndex].column = j;
			}

			matchMismatch = values.mismatch;
			if(rowBuffer.z == second[j])
				matchMismatch = values.match;

			iBuffer.curr3.y = max(iBuffer.left3.y + values.extension, iBuffer.left3.x + values.first);
			iBuffer.curr3.z = max(iBuffer.curr2.z + values.extension, iBuffer.curr2.x + values.first);
			iBuffer.curr3.x = max(max(0, iBuffer.curr3.y), max(iBuffer.curr3.z, iBuffer.left2.x + matchMismatch));

			if(iBuffer.curr3.x > score[scoreIndex].score) {
				score[scoreIndex].score = iBuffer.curr3.x;
				score[scoreIndex].row = i + 3;
				score[scoreIndex].column = j;
			}

			if(threadIdx.x < blockDim.x - 1) {
				iHbuffer[threadIdx.x + 1].x = iBuffer.curr3.x;
				iHbuffer[threadIdx.x + 1].y = iBuffer.curr3.z;
			}
			else {
				hbuffer.up[j].x = iBuffer.curr3.x;
				hbuffer.up[j].y = iBuffer.curr3.z;
			}
		}

	/*	if(threadIdx.x == tid && blockIdx.x == bl) {
			printf("blockIdx.x = %d, threadIdx.x = %d, i = %d, j = %d\n", blockIdx.x, threadIdx.x, i, j);
			printK(&iBuffer);
		}*/

		j++;

		__syncthreads();

	/*	if(threadIdx.x == tid && blockIdx.x == bl)
			printBuffers(&hbuffer, &vbuffer, iHbuffer, secondLength);*/

		int2 newUp;
		if(threadIdx.x > 0)
			newUp = iHbuffer[threadIdx.x];
		else
			newUp = hbuffer.up[j];

		pushForwardK(&iBuffer, newUp);

	/*	if(threadIdx.x == tid && blockIdx.x == bl) {
			printf("blockIdx.x = %d, threadIdx.x = %d, i = %d, j = %d\n", blockIdx.x, threadIdx.x, i, j);
			printK(&iBuffer);
			printBuffers(&hbuffer, &vbuffer, iHbuffer, secondLength);
		}*/

		__syncthreads();
	}

	if (i >= 0 && i < firstLength) {
		updateVerticalBuffer(&iBuffer, &vbuffer, i);
		if(threadIdx.x < blockDim.x -1)
			hbuffer.up[j - 1] = iHbuffer[threadIdx.x + 1];
	}

/*	if(threadIdx.x == tid && blockIdx.x == bl) {
		printBuffers(&hbuffer, &vbuffer, iHbuffer, secondLength);
	}*/
}

int main(int argc, char *argv[]) {
    printf("### Welcome to SWalign v%s\n\n", VERSION);
    cudaTimer timer;
    timer.start();

    if (argc != 7) {
        printf("Expected 6 input arguments, not %d!\n\n", argc - 1);
        return -1;
    }

    FASTAsequence first(argv[1]);
    FASTAsequence second(argv[2]);

    printf("> Loading input sequences... ");
    if(!first.load() || !second.load())
    	exitWithMsg("An error has occured while loading input sequences.", -1);
    else
    	printf("DONE\n\n");
	
    printf("First sequence of length %d:\n%s\n\n", first.getLength(), first.getSequenceName());
    printf("Second sequence of length %d:\n%s\n\n", second.getLength(), second.getSequenceName());

    printf("> Looking for CUDA capable cards... ");
    CUDAcard bestGpu = findBestDevice();
    safeAPIcall(hipSetDevice(bestGpu.cardNumber));
    printf("DONE\n\n");
    printf("Found %d CUDA capable GPU(s), picked GPU number %d:\n",
    		bestGpu.cardsInSystem, bestGpu.cardNumber + 1);
    printCardInfo(bestGpu);
    printf("\n");

    printf("> Initializing launch configuration... ");
    LaunchConfig config = getLaunchConfig(
    		min(first.getLength(), second.getLength()),
    		bestGpu);
    printf("DONE\n\n");
    printLaunchConfig(config);

    printf("\n> Preparing SWquerry... ");
    SWquerry querry(&first, &second);
    querry.prepare(config);
    printf("DONE\n\n");

    scoring values = initScoringValues(argv[3], argv[4], argv[5], argv[6]);

    printf("> Starting alignment process... ");

    alignmentScore max;
    max.score = -1;
    max.row = -1;
    max.column = -1;

    alignmentScore *score;
    int scoreSize = sizeof(alignmentScore) * config.blocks * config.threads;
    score = (alignmentScore *) malloc(scoreSize);
    if(score == NULL)
    	exitWithMsg("An error has occured while allocating blockScores array on host.", -1);
    
	alignmentScore *devScore = (alignmentScore *) cudaGetSpaceAndSet(scoreSize, 0);

    GlobalBuffer buffer;
    initGlobalBuffer(&buffer, querry.getSecond()->getPaddedLength(), config);

	int D = config.blocks + ceil(((double) querry.getFirst()->getPaddedLength())
			/ (ALPHA * config.threads)) - 1;

	safeAPIcall(hipFuncSetCacheConfig(reinterpret_cast<const void*>(shortPhase), hipFuncCachePreferShared));
	safeAPIcall(hipFuncSetCacheConfig(reinterpret_cast<const void*>(longPhase), hipFuncCachePreferShared));
	
    cudaTimer kernelTimer;
    kernelTimer.start();

    for(int dk = 0; dk < D + config.blocks; dk++) {
    	shortPhase<<<config.blocks, config.threads, config.sharedMemSize>>>(
    			dk,
    			buffer.hBuffer,
    			buffer.vBuffer,
    			querry.getDevFirst(),
    			querry.getFirst()->getPaddedLength(),
    			querry.getDevSecond(),
    			querry.getSecond()->getPaddedLength(),
    			values,
    			devScore
    			);

    	safeAPIcall(hipDeviceSynchronize());
		longPhase<<<config.blocks, config.threads, config.sharedMemSize>>>(
				dk,
    			buffer.hBuffer,
    			buffer.vBuffer,
    			querry.getDevFirst(),
    			querry.getFirst()->getPaddedLength(),
    			querry.getDevSecond(),
    			querry.getSecond()->getPaddedLength(),
				values,
				devScore
				);

		safeAPIcall(hipDeviceSynchronize());
    }
    
    kernelTimer.stop();

    safeAPIcall(hipMemcpy(score, devScore, scoreSize, hipMemcpyDeviceToHost));
	for(int i = 0; i < config.blocks * config.threads; i++) {
		if(max.score < score[i].score) {
			max.score = score[i].score;
			max.column = score[i].column;
			max.row = score[i].row;
		}
    }

	timer.stop();

    printf("DONE\n\n");

    printf("Kernel executed in %f s\n", kernelTimer.getElapsedTimeMillis() / 1000);
    printf("Application executed in %f s\n", timer.getElapsedTimeMillis() / 1000);

    printf("\nAlignment score: %d at [%d, %d]\n", max.score, max.row + 1, max.column + 1);

    safeAPIcall(hipFree(devScore));

    freeGlobalBuffer(&buffer);

    free(score);

    return 0;
}
