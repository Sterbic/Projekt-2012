#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdlib.h>

#include "FASTA.h"

#define MAXLEN 5005
#define VERSION "0.2.0"

typedef struct {
    int match;
    int mismatch;
    int insertion;
    int deletion;
} scoring;

typedef struct {
    int score;
    int row;
    int column;
} entry;

__global__ void kernel() {

}

int main(int argc, char *argv[]) {
	printf("> Welcome to SWaligner v%s\n\n", VERSION);

	if (argc != 7) {
        printf("Expected 6 input arguments, not %d!\n\n", argc - 1);
        return -1;
    }

	FASTAsequence first(argv[1]);
	FASTAsequence second(argv[2]);

	printf("Loading input sequences... ");
	if(!first.load() || !second.load()) {
		printf("ERROR\nAn error has occured while loading input sequences!\n\n");
		exit(-1);
	}
	else {
		printf("DONE\n\n");
	}

	printf("First sequence:\n%s\n\n", first.getSequenceName());
	printf("Second sequence:\n%s\n\n", second.getSequenceName());

	scoring values;
	printf("Initializing scoring values... ");
    values.match = atoi(argv[3]);
    values.mismatch = atoi(argv[4]);
    values.insertion = atoi(argv[5]);
    values.deletion = atoi(argv[6]);

	if(values.match < 1 || values.mismatch > -1 || values.insertion > -1 || values.deletion > -1) {
		printf("ERROR\nOne or more scoring values were not usable!\n\n");
		exit(-1);
	}
	else {
		printf("DONE\n\nScoring values:\n");
		printf("	>Match: %d\n", values.match);
		printf("	>Mismatch: %d\n", values.mismatch);
		printf("	>Insertion: %d\n", values.insertion);
		printf("	>Deletion: %d\n\n", values.deletion);
	}

	printf("Starting alignment process... ");

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	kernel<<<10, 10>>>();

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	printf("DONE\n\n");

	float time;
	hipEventElapsedTime(&time, start, stop);
	printf("Kernel executed in %f s\n", time / 1000);

    return 0;
}
